#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

#define BLOCK_SIZE 256

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__
        __global__ void kernNaiveScanPass(int n, int offset, const int* in, int* out) {
            int i = threadIdx.x + blockIdx.x * blockDim.x;
            if (i >= n) return;

            if (i >= offset) {
                out[i] = in[i] + in[i - offset];
            }
            else {
                out[i] = in[i];
            }
        }

        __global__ void kernInclusiveToExclusive(int n, const int* in, int* out) {
            int i = threadIdx.x + blockIdx.x * blockDim.x;
            if (i >= n) return;

            out[i] = (i == 0) ? 0 : in[i - 1];
        }

        void scanDevice(int n, int* dev_out, const int* dev_in) {
            int* dev_bufA = nullptr;
            int* dev_bufB = nullptr;

            hipMalloc(&dev_bufA, n * sizeof(int));
            hipMalloc(&dev_bufB, n * sizeof(int));
            hipMemcpy(dev_bufA, dev_in, n * sizeof(int), hipMemcpyHostToDevice);

            int gridSize = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

            int passes = ilog2ceil(n);

            int* in = dev_bufA;
            int* out = dev_bufB;

            for (int i = 0; i < passes; i++) {
                int offset = 1 << i;
                kernNaiveScanPass<<<gridSize, BLOCK_SIZE >>>(n, offset, in, out);
                hipDeviceSynchronize();
                std::swap(in, out);
            }
            kernInclusiveToExclusive<<<gridSize, BLOCK_SIZE >>>(n, in, out);
            hipDeviceSynchronize();

            hipMemcpy(dev_out, out, n * sizeof(int), hipMemcpyDeviceToDevice);

            hipFree(dev_bufA);
            hipFree(dev_bufB);
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            if (n <= 0) {
                return;
            }
            int* dev_in, * dev_out;
            hipMalloc(&dev_in, n * sizeof(int));
            hipMalloc(&dev_out, n * sizeof(int));
            hipMemcpy(dev_in, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();
            scanDevice(n, dev_out, dev_in);
            timer().endGpuTimer();

            hipMemcpy(odata, dev_out, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_in);
            hipFree(dev_out);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int* odata, const int* idata) {
            int* dev_in, * dev_flags, * dev_indices, * dev_out;
            hipMalloc(&dev_in, n * sizeof(int));
            hipMalloc(&dev_flags, n * sizeof(int));
            hipMalloc(&dev_indices, n * sizeof(int));
            hipMalloc(&dev_out, n * sizeof(int));

            hipMemcpy(dev_in, idata, n * sizeof(int), hipMemcpyHostToDevice);

            int gridSize = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

            timer().startGpuTimer();

            timer().startGpuSubTimer("map");
            StreamCompaction::Common::kernMapToBoolean<<<gridSize, BLOCK_SIZE >>>(n, dev_flags, dev_in);
            hipDeviceSynchronize();
            timer().endGpuSubTimer();

            timer().startGpuSubTimer("scan");
            scanDevice(n, dev_indices, dev_flags);
            hipDeviceSynchronize();
            timer().endGpuSubTimer();

            int lastScan, lastFlag;
            hipMemcpy(&lastScan, dev_indices + (n - 1), sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&lastFlag, dev_flags + (n - 1), sizeof(int), hipMemcpyDeviceToHost);
            int validCount = lastScan + lastFlag;

            timer().startGpuSubTimer("scatter");
            StreamCompaction::Common::kernScatter<<<gridSize, BLOCK_SIZE >>>(n, dev_out, dev_in, dev_flags, dev_indices);
            hipDeviceSynchronize();
            timer().endGpuSubTimer();

            timer().endGpuTimer();

            if (validCount > 0) {
                hipMemcpy(odata, dev_out, validCount * sizeof(int), hipMemcpyDeviceToHost);
            }

            hipFree(dev_in);
            hipFree(dev_flags);
            hipFree(dev_indices);
            hipFree(dev_out);

            return validCount;
        }
    }
}
